#include "hip/hip_runtime.h"
/*==============================================================================
 *$RCSfile: bmark_lite.c,v $
 *
 *   DESC : This file contains the Test Main and other TH support functions
 *
 * AUTHOR : ARM, Ltd., Rick Foos, ECL, LLC
 *
 *  EEMBC : Automotive Subcommittee 
 *
 *    CVS : $Revision: 1.15 $
 *          $Date: 2002/08/07 22:20:41 $
 *          $Author: rick $
 *          $Source: d:/cvs/eembc2/automotive/aiifft01/bmark_lite.c,v $
 *          
 * NOTE   :
 *
 *------------------------------------------------------------------------------
 *
 * HISTORY :
 *
 * $Log: bmark_lite.c,v $
 * Revision 1.15  2002/08/07 22:20:41  rick
 * Add NI CRC to TH Regular
 *
 * Revision 1.14  2002/07/22 21:59:12  rick
 * General cleanup Beta 2b
 *
 * Revision 1.13  2002/07/22 16:10:04  rick
 * Fix iteration dependant NI CRC
 *
 * Revision 1.12  2002/07/18 23:33:49  rick
 * Fix iteration dependencies in NI CRC
 *
 * Revision 1.11  2002/07/10 19:00:50  rick
 * Always initialize tcdef->CRC
 *
 * Revision 1.10  2002/05/29 22:24:53  rick
 * Set recommended iterations with make
 *
 * Revision 1.9  2002/05/10 23:57:45  rick
 * Fix missing shifts in 16, and 32 bit CRC calculations
 *
 * Revision 1.8  2002/05/10 17:20:36  rick
 * Add al_main to API
 *
 * Revision 1.7  2002/04/25 20:10:43  rick
 * sprintf to th_sprintf
 *
 * Revision 1.6  2002/04/10 19:53:19  rick
 * Fix CRC error
 *
 * Revision 1.5  2002/04/10 19:36:58  rick
 * Fixes to reduce Lite vs. Regular variances in timing
 *
 * Revision 1.4  2002/03/11 22:11:47  rick
 * ITERATIONS, CRC_CHECK, NON_INTRUSIVE TCDef Usage
 *
 * Revision 1.3  2002/02/25 17:15:32  rick
 * Add comment blocks, fix atime th_report call.
 *
 *   5/14/99 pjt     1.0
 *       Initial cleanup
 *   7/4/99  arw     1.0
 *       Added eembc_dt.h datatypes & additional cleanup
 *       "thlib.h" will #include eembc_dt.h for us
 *	7/13/99 arw		1.0
 *		Corrected WriteOut() and 
 *		Changed variable names starting with debugOut* to RAMfile*
 *		Changed various declarations
 *   5/29/01 TC 
 *       Removed all debug related code to debug.c
 *       Removed all intializatio and variable declaration to init.c
 *       Added all extern in algo.h
 *       Removed 'varsize', mostly replaced with n_int 
 *       Combined THResult structure and TCDef structure into one	
 *       Added CRC in TCDef structure - Calculates running CRC of all the
 *       intermediate value
 *       added #if CRC_CHECK, so that number of iterations can be changed
 *       In order to match CRC, you have to use pre-defined number of iterations
 *       Removed all dynamic memory allocation
 *   7/24/01 TC/arw  8/16 Bit Benchmark Suite (V 2.0)
 *       Added optional compile define for all CRC operation with CRC_CHECK.
 *       turn on CRC operation by setting #define CRC_CHECK	TRUE
 *       turn off CRC operation by setting #define CRC_CHECK FALSE 
 *       define for CRC_CHECK is in th\al\thcfg.h file
 *       Please use recommended # of iterations when CRC_CHECK is enabled
 *
 *------------------------------------------------------------------------------
 * Copyright (c) 1998-2002 by the EDN Embedded Microprocessor 
 * Benchmark Consortium (EEMBC), Inc. 
 * 
 * All Rights Reserved. This is licensed program product and 
 * is owned by EEMBC. The Licensee understands and agrees that the 
 * Benchmarks licensed by EEMBC hereunder (including methods or concepts 
 * utilized therein) contain certain information that is confidential 
 * and proprietary which the Licensee expressly agrees to retain in the 
 * strictest confidence and to use only in conjunction with the Benchmarks 
 * pursuant to the terms of this Agreement. The Licensee further agrees 
 * to keep the source code and all related documentation confidential and 
 * not to disclose such source code and/or related documentation to any 
 * third party. The Licensee and any READER of this code is subject to 
 * either the EEMBC Member License Agreement and/or the EEMBC Licensee 
 * Agreement. 
 * TO THE MAXIMUM EXTENT PERMITTED BY APPLICABLE LAW, EEMBC DISCLAIMS ALL 
 * WARRANTIES, EITHER EXPRESS OR IMPLIED, INCLUDING, BUT NOT LIMITED TO, 
 * IMPLIED WARRANTIES OF MERCHANTABILITY OR FITNESS FOR A PARTICULAR 
 * PURPOSE, WITH REGARD TO THE BENCHMARKS AND THE ACCOMPANYING 
 * DOCUMENTATION. LICENSEE ACKNOWLEDGES AND AGREES THAT THERE ARE NO 
 * WARRANTIES, GUARANTIES, CONDITIONS, COVENANTS, OR REPRESENTATIONS BY 
 * EEMBC AS TO MARKETABILITY, FITNESS FOR A PARTICULAR PURPOSE, OR OTHER 
 * ATTRIBUTES, WHETHER EXPRESS OR IMPLIED (IN LAW OR IN FACT), ORAL OR 
 * WRITTEN. 
 * 
 * Licensee hereby agrees by accessing this source code that all benchmark 
 * scores related to this code must be certified by ECL prior to publication 
 * in any media, form, distribution, or other means of conveyance of 
 * information subject to the terms of the EEMBC Member License Agreement 
 * and/or EEMBC Licensee Agreement. 
 * 
 * Other Copyright Notice (if any): 
 * 
 * For conditions of distribution and use, see the accompanying README file.
 * ===========================================================================*/

/*******************************************************************************
    Includes                                                                    
*******************************************************************************/

#define ALGO_GLOBALS    1   /* Next time, we'll skip these */
#include "algo.h"

/* Estimate of allocation for NUM_TESTS*( debug test + 2 variables )*/
#define T_BSIZE (MAX_FILESIZE+((NUM_TESTS+1)*VAR_COUNT*4))

#define NUM_POINTS    512   /* Number of points in FFT */
#define FFT_LENGTH    9     /* length = log2( NUM_POINTS )*/

#ifdef PI
#undef PI
#endif
#define PI    3.141592654

/* TRIG_SCALE_FACTOR is used for greatest twiddle factor precision */
#define TRIG_SCALE_FACTOR           32768

/* BUTTERFLY_SCALE_FACTOR = log2( TRIG_SCALE_FACTOR )is used to
accomodate accumulator size limit of 32 bits */
#define BUTTERFLY_SCALE_FACTOR      15

/* STAGE_SCALE_FACTOR is used to prevent overflow from
accumulation at each processing stage */
#define STAGE_SCALE_FACTOR          1

int NUMBEROFTEST =1024;

typedef struct{
    varsize realData_1[NUM_POINTS] ;    /* Points to real part of data */
    varsize imagData_1[NUM_POINTS] ;    /* Points to imaginary part of data */
}inputData;

typedef struct{

}outputData;

/* ======================================================================== */
/*         F U N C T I O N   P R O T O T Y P E S                            */
/* ======================================================================== */
__device__ void aiifft01( varsize realData_1[NUM_POINTS], varsize imagData_1[NUM_POINTS]);


__device__ void aiifft01( varsize realData_1[NUM_POINTS], varsize imagData_1[NUM_POINTS])
{
    int bitRevInd[NUM_POINTS] ;
    double trigArg ; 
    int index ;
    int brIndex ;
     varsize sineV[NUM_POINTS / 2] ;
     varsize cosineV[NUM_POINTS / 2] ;
     varsize realBitRevData_1[NUM_POINTS] ;
     varsize imagBitRevData_1[NUM_POINTS] ;
     varsize wReal_1 ;
     varsize wImag_1 ;
     varsize tRealData_1 ;
     varsize tImagData_1 ;
     varsize *realLow_1 ;
     varsize *imagLow_1 ;
     varsize *realHi_1 ;
     varsize *imagHi_1 ;
     long argIndex_1 ;
     long deltaIndex_1 ;
     varsize n1_1 ;
     varsize n2_1 ;
     varsize l_1 ;
     varsize i_1 ;
     varsize j_1 ;
     varsize k_1 ;
     varsize passCount_1 ;
    /* Populate the sine & cosine tables -- used by all instances */    
    for( i_1 = 0 ; i_1 < ( NUM_POINTS / 2 ) ; i_1++ )
    {        
        trigArg = (double)( i_1 * PI / ( NUM_POINTS / 2 ) ) ; 

        if( sin( trigArg ) == 1.0 )
        {
            sineV[i_1] = (long)( TRIG_SCALE_FACTOR - 1 ) ;
        }
        else
        {

            sineV[i_1] = (long)( sin( trigArg ) * TRIG_SCALE_FACTOR ) ;
        }

        if( cos( trigArg ) == 1.0 )
        {
            cosineV[i_1] = (long)( TRIG_SCALE_FACTOR - 1 ) ;
        }
        else
        {
            cosineV[i_1] = (long)( cos( trigArg ) * TRIG_SCALE_FACTOR ) ;
        }
    }

    /* Compute the bit reversal indicies  -- used by all the instances */    
    for( i_1 = 0 ; i_1 < NUM_POINTS ; i_1++ )
    {        
        index = i_1 ; 
        brIndex = 0 ; 
        for( j_1 = 0 ; j_1 < FFT_LENGTH ; j_1++ )
        {        
            brIndex <<= 1 ; 
            if( 0x01 &index )
            {
                brIndex |= 0x01 ; 
            }
            index >>= 1 ; 
        }
        bitRevInd[i_1] = brIndex ; 
    }

    /*
     * This is the actual benchmark algorithm.
     *
     */    

    /*
     * Inverse FFT processing in the automotive area might be applied in audio
     * signal processing, specifically noise-cancellation processes.
     *
     * Compute the Radix 2 Decimation in Frequency iFFT on the complex input
     * values stored in the 'realData' and 'imagData' arrays.  Converts
     * frequency-domain data to time-domain.  Builds the sine and cosine
     * twiddle factors prior to execution of the iFFT loop.  Also builds 
     * the bit reversal indicies prior to execution of the loop.
     *
     */

        /***********************************************************************
            First Pass                                                          
        ***********************************************************************/

        /* Bit Reversal */    
        for( i_1 = 0 ; i_1 < NUM_POINTS ; i_1++ )
        {        
            realBitRevData_1[i_1] = realData_1[bitRevInd[i_1]] ; 
            imagBitRevData_1[i_1] = imagData_1[bitRevInd[i_1]] ; 
        }

        /* Return bit reversed data to input arrays */    
        for( i_1 = 0 ; i_1 < NUM_POINTS ; i_1++ )
        {        
            realData_1[i_1] = realBitRevData_1[i_1] ; 
            imagData_1[i_1] = imagBitRevData_1[i_1] ; 
        }

        /* iFFT Computation */    

        /* Step through the stages */    

        for( passCount_1 = 0, k_1 = 1 ; k_1 <= FFT_LENGTH ; k_1++, passCount_1++ )
        {        
            n1_1 = 1 << k_1 ; 
            n2_1 = n1_1 >> 1 ; 

            /* Initialize twiddle factor lookup indicies */    
            argIndex_1 = 0 ; 
            deltaIndex_1 = ( NUM_POINTS / 2 ) / n2_1 ; 

            /* Step through the butterflies */    
            for( j_1 = 0 ; j_1 < n2_1 ; j_1++, passCount_1++ )
            {        
                /* Lookup twiddle factors */    
                wReal_1 = cosineV[argIndex_1] ; 
                /* Note iFFT reversal of sign */
                wImag_1 = -sineV[argIndex_1] ;

                /* Process butterflies with the same twiddle factors */    
                for( i_1 = j_1 ; i_1 < NUM_POINTS ; i_1 += n1_1, passCount_1++ )
                {
                    l_1 = i_1 + n2_1 ; 
                    realLow_1 = &realData_1[l_1] ; 
                    imagLow_1 = &imagData_1[l_1] ; 
                    realHi_1 = &realData_1[i_1] ; 
                    imagHi_1 = &imagData_1[i_1] ; 

                    /* Scale each stage to prevent overflow */    
                    *realLow_1 >>= STAGE_SCALE_FACTOR ; 
                    *imagLow_1 >>= STAGE_SCALE_FACTOR ; 
                    *realHi_1 >>= STAGE_SCALE_FACTOR ; 
                    *imagHi_1 >>= STAGE_SCALE_FACTOR ; 

                    tRealData_1 = *realLow_1 * wReal_1 - *imagLow_1 * wImag_1 ; 
                    tImagData_1 = *imagLow_1 * wReal_1 + *realLow_1 * wImag_1 ; 

                    /* Scale twiddle products to accomodate 32-bit accu. */
                    tRealData_1 >>= BUTTERFLY_SCALE_FACTOR ; 
                    tImagData_1 >>= BUTTERFLY_SCALE_FACTOR ; 

                    realData_1[l_1] = *realHi_1 - tRealData_1 ; 
                    imagData_1[l_1] = *imagHi_1 - tImagData_1 ; 
                    realData_1[i_1] += tRealData_1 ; 
                    imagData_1[i_1] += tImagData_1 ; 
                }
                argIndex_1 += deltaIndex_1 ; 
            }
        } /* End of iFFT loop */
}

void readFile(inputData* data){
    int index=0;
    int i=0;
    int x,y;
    char c;
    int ret = fscanf(stdin, "%c %ld %ld ", &c, &x, &y);
    while(ret != EOF){
        while(i<NUM_POINTS){
            data[index].realData_1[i] = x;
            data[index].imagData_1[i] = y;
            i++;
            if(i!=NUM_POINTS)
                ret = fscanf(stdin, "%ld %ld ", &x, &y);
            else
                ret = fscanf(stdin, "%c %ld %ld ", &c, &x, &y);
        }
        i=0;
        index++;
    }
    printf("Index %d\n", index);
    NUMBEROFTEST = index;
}

__global__ void myKernel(inputData* data, outputData* outData){
    int index =  blockIdx.x * blockDim.x + threadIdx.x;
    aiifft01(data[index].realData_1,data[index].imagData_1);

}

void launch(inputData *dataCPU, outputData* outData, char* fileName){
    int threadsPerBlock = 32;
    int blocksPerGrid = NUMBEROFTEST / threadsPerBlock;
    int out_size = NUMBEROFTEST * sizeof(outputData);
    int input_size = NUMBEROFTEST * sizeof(inputData);
    inputData *dataGPUIn;
    outputData* dataGPUOut;
    hipMalloc ((void**)&dataGPUIn, input_size);
    hipMemcpy(dataGPUIn, dataCPU, input_size, hipMemcpyHostToDevice );
    hipMalloc ((void**)&dataGPUOut, out_size);
    hipMemcpy(dataGPUOut, outData, out_size, hipMemcpyHostToDevice );
    myKernel << <blocksPerGrid, threadsPerBlock >> >(dataGPUIn, dataGPUOut);
    hipMemcpy(outData, dataGPUOut, out_size, hipMemcpyDeviceToHost);
    hipFree(dataGPUIn);
    hipFree(dataGPUOut);
}

/***************************************************************************/
int main(int argc, char* argv[] )
{
    inputData* data = (inputData*)malloc(sizeof(inputData) * NUMBEROFTEST);
    outputData* outD = (outputData*)malloc(sizeof(outputData) * NUMBEROFTEST);
    readFile(data);
    launch(data, outD, argv[1]);
    for(int i=0; i<NUMBEROFTEST; i++){
        fprintf(stdout, "ID# %d: ", i);
        for( int i_1 = 0 ; i_1 < NUM_POINTS ; i_1++ )
        {
            fprintf(stdout, " %ld ", data[i].realData_1[i_1] ) ;
        }
        fprintf(stdout, "\n");
    }
    free(data);
    free(outD);
    return 0;
} 

