#include "hip/hip_runtime.h"
/*==============================================================================
 *$RCSfile: bmark_lite.c,v $
 *
 *   DESC : This file contains the Test Main and other TH support functions
 *
 * AUTHOR : ARM, Ltd., Rick Foos, ECL, LLC
 *
 *  EEMBC : Automotive Subcommittee 
 *
 *    CVS : $Revision: 1.17 $
 *          $Date: 2002/08/07 22:20:35 $
 *          $Author: rick $
 *          $Source: d:/cvs/eembc2/automotive/aifftr01/bmark_lite.c,v $
 *          
 * NOTE   :
 *
 *------------------------------------------------------------------------------
 *
 * HISTORY :
 *
 * $Log: bmark_lite.c,v $
 * Revision 1.17  2002/08/07 22:20:35  rick
 * Add NI CRC to TH Regular
 *
 * Revision 1.16  2002/07/22 21:59:08  rick
 * General cleanup Beta 2b
 *
 * Revision 1.15  2002/07/22 16:09:57  rick
 * Fix iteration dependant NI CRC
 *
 * Revision 1.14  2002/07/18 23:33:47  rick
 * Fix iteration dependencies in NI CRC
 *
 * Revision 1.13  2002/07/10 19:00:48  rick
 * Always initialize tcdef->CRC
 *
 * Revision 1.12  2002/05/29 22:24:49  rick
 * Set recommended iterations with make
 *
 * Revision 1.11  2002/05/10 23:57:44  rick
 * Fix missing shifts in 16, and 32 bit CRC calculations
 *
 * Revision 1.10  2002/05/10 17:20:36  rick
 * Add al_main to API
 *
 * Revision 1.9  2002/04/25 20:10:43  rick
 * sprintf to th_sprintf
 *
 * Revision 1.8  2002/04/19 22:17:01  rick
 * Code cleanup for Linux build
 *
 * Revision 1.7  2002/04/10 20:02:52  rick
 * Fix CRC values
 *
 * Revision 1.6  2002/04/10 19:36:56  rick
 * Fixes to reduce Lite vs. Regular variances in timing
 *
 * Revision 1.5  2002/03/11 22:11:47  rick
 * ITERATIONS, CRC_CHECK, NON_INTRUSIVE TCDef Usage
 *
 * Revision 1.4  2002/02/25 17:15:31  rick
 * Add comment blocks, fix atime th_report call.
 *
 *
 *------------------------------------------------------------------------------
 * Copyright (c) 1998-2002 by the EDN Embedded Microprocessor 
 * Benchmark Consortium (EEMBC), Inc. 
 * 
 * All Rights Reserved. This is licensed program product and 
 * is owned by EEMBC. The Licensee understands and agrees that the 
 * Benchmarks licensed by EEMBC hereunder (including methods or concepts 
 * utilized therein) contain certain information that is confidential 
 * and proprietary which the Licensee expressly agrees to retain in the 
 * strictest confidence and to use only in conjunction with the Benchmarks 
 * pursuant to the terms of this Agreement. The Licensee further agrees 
 * to keep the source code and all related documentation confidential and 
 * not to disclose such source code and/or related documentation to any 
 * third party. The Licensee and any READER of this code is subject to 
 * either the EEMBC Member License Agreement and/or the EEMBC Licensee 
 * Agreement. 
 * TO THE MAXIMUM EXTENT PERMITTED BY APPLICABLE LAW, EEMBC DISCLAIMS ALL 
 * WARRANTIES, EITHER EXPRESS OR IMPLIED, INCLUDING, BUT NOT LIMITED TO, 
 * IMPLIED WARRANTIES OF MERCHANTABILITY OR FITNESS FOR A PARTICULAR 
 * PURPOSE, WITH REGARD TO THE BENCHMARKS AND THE ACCOMPANYING 
 * DOCUMENTATION. LICENSEE ACKNOWLEDGES AND AGREES THAT THERE ARE NO 
 * WARRANTIES, GUARANTIES, CONDITIONS, COVENANTS, OR REPRESENTATIONS BY 
 * EEMBC AS TO MARKETABILITY, FITNESS FOR A PARTICULAR PURPOSE, OR OTHER 
 * ATTRIBUTES, WHETHER EXPRESS OR IMPLIED (IN LAW OR IN FACT), ORAL OR 
 * WRITTEN. 
 * 
 * Licensee hereby agrees by accessing this source code that all benchmark 
 * scores related to this code must be certified by ECL prior to publication 
 * in any media, form, distribution, or other means of conveyance of 
 * information subject to the terms of the EEMBC Member License Agreement 
 * and/or EEMBC Licensee Agreement. 
 * 
 * Other Copyright Notice (if any): 
 * 
 * For conditions of distribution and use, see the accompanying README file.
 * ===========================================================================*/

/*******************************************************************************
    Includes                                                                    
*******************************************************************************/
#define ALGO_GLOBALS    1   /* Next time, we'll skip these */
#include "algo.h"

/* Estimate of allocation for NUM_TESTS*( debug test + 2 variables )*/
#define T_BSIZE (MAX_FILESIZE+((NUM_TESTS+1)*VAR_COUNT*4))
int NUMBEROFTEST =1024;
#define STAGE_SCALE_FACTOR      1
#define NUM_POINTS   512
#define BUTTERFLY_SCALE_FACTOR  15
#define PI 3.141592654 
#define TRIG_SCALE_FACTOR       32768
#define FFT_LENGTH   9 
/* ======================================================================== */
/*         F U N C T I O N   P R O T O T Y P E S                            */
/* ======================================================================== */
__device__ void t_run_test(struct TCDef *tcdef, int argc, const char* argv[]);

typedef struct{
    varsize realData_1[NUM_POINTS] ;    /* Points to real part of data */
    varsize imagData_1[NUM_POINTS] ;    /* Points to imaginary part of data */
}inputData;
typedef struct{
    varsize result;
}outputData;


/*******************************************************************************
    Local Data                                                                  
*******************************************************************************/

/* >> IMPORTANT NOTE << 
* Since benchmarks can be entered( run )multiple times, the benchmark
* MUST NOT depend on global data being initialized.  E.g. it must
* complelty initialize itself EVERY TIME its t_run_test()function
* is called.
* 
*/    

/*********************************************************************************
* FUNC	: int t_run_test( struct TCDef *tcdef,int argc, const char *argv[] )
* DESC  : This is the functions that carries out the algorithm. This function
*         is called from the main.  Same algorithm is called three times. Input
*         data for the algorithm is stored in 'algotst'c'.  All variables declared
*         and initialized in 'init.c'.  When 'BMDEBUG' and 'WINDOWS_EXAMPLE_CODE' 
*         defined in 'thcfg.h' it will write bunch of debug message in the RAM.  
*         All debug routines are in 'debug.c'. It calculates CRC using intermediate
*         values from the algorithms.  CRC is used to check if algorithm carried out
*         correctly.
* ARGUMENT: arc - not used
*           argv - not used
*           tcdef - structure defined in 'thlib.h'.  Following members of tcdef
*     			rec_iterations- recommend number of times algorithm should be carried out
*   			iterations  - actual number of times algorithm was carried
*                             most cases same as 'rec_iterations'
*               duration	- Total time to carry out the algorithm
*               CRC			- calculated CRC
*
* RETURNS : Success if the the CRC matches. 
*****************************************************************************************/    
__device__ void t_run_test(varsize realData_1[NUM_POINTS], varsize imagData_1[NUM_POINTS])    /* Points to imaginary part of data */
{
    /* Input test data table looped */
    int bitRevInd[NUM_POINTS] ; 
    double trigArg ; 
    int index ;
    int brIndex ; 
    varsize sineV[NUM_POINTS / 2] ;
    varsize cosineV[NUM_POINTS / 2] ;
    varsize realBitRevData_1[NUM_POINTS] ;
    varsize imagBitRevData_1[NUM_POINTS] ;
    varsize wReal_1 ;
    varsize wImag_1 ;
    varsize tRealData_1;
    varsize tImagData_1 ;
    varsize *realLow_1 ;
    varsize *imagLow_1 ;
    varsize *realHi_1 ;
    varsize *imagHi_1 ;
    long argIndex_1 ;
    long deltaIndex_1;
    varsize n1_1 ;
    varsize n2_1 ;
    varsize l_1 ;
    varsize i_1 ;
    varsize j_1 ;
    varsize k_1; 
    varsize passCount_1;

    /*
     * First, initialize the data structures we need for the test
     * and allocate memory as needed.  Report an error if we can't.
     *
     */    

    /* Variable initializations at t=0 */    
		/* 
		 * BUG  145 in v1, realdata_2 is loaded, and unused. Pass 1 data is invalid
		 * If workloads unchanged, 
		 * Temporary fix should stabilize ports which do not initialize
		 * globals to 0 by default. VC performance results should be unchanged.
		 * for V2: isTableLooped = GetInputValues( realData_1, imagData_1 ) ;
		 * for V1.1: set realData_1 and imagData_1 to 0.
		 * 
		 */


    /* Populate the sine & cosine tables -- used by all instances */    
    for( i_1 = 0 ; i_1 < ( NUM_POINTS / 2 ) ; i_1++ )
    {        
        trigArg = (double)( i_1 * PI / ( NUM_POINTS / 2 ) ) ; 

        if( sin( trigArg ) == 1.0 )
        {
            sineV[i_1] = (long)( TRIG_SCALE_FACTOR - 1 ) ; 
        }
        else
        {

            sineV[i_1] = (long)( sin( trigArg ) * TRIG_SCALE_FACTOR ) ; 
        }

        if( cos( trigArg ) == 1.0 )
        {
            cosineV[i_1] = (long)( TRIG_SCALE_FACTOR - 1 ) ; 
        }
        else
        {
            cosineV[i_1] = (long)( cos( trigArg ) * TRIG_SCALE_FACTOR ) ; 
        }
    }

    /* Compute the bit reversal indicies  -- used by all the instances */    
    for( i_1 = 0 ; i_1 < NUM_POINTS ; i_1++ )
    {        
        index = i_1 ; 
        brIndex = 0 ; 
        for( j_1 = 0 ; j_1 < FFT_LENGTH ; j_1++ )
        {        
            brIndex <<= 1 ; 
            if( 0x01 &index )
            {
                brIndex |= 0x01 ; 
            }
            index >>= 1 ; 
        }
        bitRevInd[i_1] = brIndex ; 
    }

    /*
     * This is the actual benchmark algorithm.
     *
     */    

    /* The FFT is a basic signal-processing function which can be applied to 
     * many test suites. Therefore, it is important to create an FFT benchmark
     * in the Automotive/Industrial suite which is representative of
     * "real-world" applications.  FFT processing in the automotive area 
     * might be applied in audio signal processing, specifically 
     * noise-cancellation processes, or speech recognition.  FFT in the 
     * automotive realm might also be used in signal processing for 
     * anti-collision systems.  FFT for industrial applications runs the 
     * gamut of resolution and frequency range.
     *
     * Compute the Radix 2 Decimation in Frequency iFFT on the complex input
     * values stored in the 'realData' and 'imagData' arrays.  Converts 
     * frequency-domain data to time-domain. Builds the sine and cosine
     * twiddle factors prior to execution of the iFFT loop. Also builds the
     * bit reversal indicies prior to execution of the loop. First, we do 
     * FFT, then compute power spectrum.
     *
     * This FFT is different than that used in the Telecom subcommittee.
     *
     */    


        /***********************************************************************
            First Pass                                                          
        ***********************************************************************/

        /* Gets 'signal_in' value from test data*/
		/* 
		 * BUG  145 in v1, realdata_2 is loaded, and unused. Pass 1 data is invalid
		 * If workloads unchanged, 
		 * Temporary fix should stabilize ports which do not initialize
		 * globals to 0 by default. VC performance results should be unchanged.
		 * for V2: isTableLooped = GetInputValues( realData_1, imagData_1 ) ;
		 * for V1.1: set realData_1 and imagData_1 to 0.
		 * 
		 */


        /* Bit Reversal */    
        for( i_1 = 0 ; i_1 < NUM_POINTS ; i_1++ )
        {        
            realBitRevData_1[i_1] = realData_1[bitRevInd[i_1]] ; 
            imagBitRevData_1[i_1] = imagData_1[bitRevInd[i_1]] ; 
        }

        /* Return bit reversed data to input arrays */    
        for( i_1 = 0 ; i_1 < NUM_POINTS ; i_1++ )
        {        
            realData_1[i_1] = realBitRevData_1[i_1] ; 
            imagData_1[i_1] = imagBitRevData_1[i_1] ; 
        }

        /* FFT Computation */    

        /* Step through the stages */    
        for( passCount_1 = 0, k_1 = 1 ; k_1 <= FFT_LENGTH ; k_1++, passCount_1++ )
        {        
            n1_1 = 1 << k_1 ; 
            n2_1 = n1_1 >> 1 ; 
             
            /* Initialize twiddle factor lookup indicies */    
            argIndex_1 = 0 ; 
            deltaIndex_1 = ( NUM_POINTS / 2 ) / n2_1 ; 

            /* Step through the butterflies */    
            for( j_1 = 0 ; j_1 < n2_1 ; j_1++, passCount_1++ )
            {        
                /* Lookup twiddle factors */    
                wReal_1 = cosineV[argIndex_1] ; 
                wImag_1 = sineV[argIndex_1] ; 

                /* Process butterflies with the same twiddle factors */    
                for( i_1 = j_1 ; i_1 < NUM_POINTS ; i_1 += n1_1, passCount_1++ )
                {        
                    l_1 = i_1 + n2_1 ; 
                    realLow_1 = &realData_1[l_1] ; 
                    imagLow_1 = &imagData_1[l_1] ; 
                    realHi_1 = &realData_1[i_1] ; 
                    imagHi_1 = &imagData_1[i_1] ; 

                    /* Scale each stage to prevent overflow */    
                    *realLow_1 >>= STAGE_SCALE_FACTOR ; 
                    *imagLow_1 >>= STAGE_SCALE_FACTOR ; 
                    *realHi_1 >>= STAGE_SCALE_FACTOR ; 
                    *imagHi_1 >>= STAGE_SCALE_FACTOR ; 

                    tRealData_1 = *realLow_1 * wReal_1 - *imagLow_1 * wImag_1 ; 
                    tImagData_1 = *imagLow_1 * wReal_1 + *realLow_1 * wImag_1 ; 

                    /* Scale twiddle products to accomodate 
                     * 32bit accumulator */
                    tRealData_1 >>= BUTTERFLY_SCALE_FACTOR ; 
                    tImagData_1 >>= BUTTERFLY_SCALE_FACTOR ; 

                    realData_1[l_1] = *realHi_1 - tRealData_1 ; 
                    imagData_1[l_1] = *imagHi_1 - tImagData_1 ; 
                    realData_1[i_1] += tRealData_1 ; 
                    imagData_1[i_1] += tImagData_1 ; 
                }
                argIndex_1 += deltaIndex_1 ; 
            }

        } /* End of FFT loop */

        /* Compute power spectrum */    
        for( i_1 = 0 ; i_1 < NUM_POINTS ; i_1++ )
        {        
            /* This can all easily overflow! */
            realData_1[i_1] *= realData_1[i_1] ;
            realData_1[i_1] += imagData_1[i_1] * imagData_1[i_1] ; 
        }
} 

void readFile(inputData* data){
    // printf("READ FILE");

    int index=0;
    int i=0;
    int x,y;
    char c;
    int ret = fscanf(stdin, "%c %ld %ld ", &c, &x, &y);
    while(ret != EOF){
        while(i<NUM_POINTS){
            data[index].realData_1[i] = x;
            data[index].imagData_1[i] = y;
            i++;
            if(i!=NUM_POINTS)
            	ret = fscanf(stdin, "%ld %ld ", &x, &y);
            else
		ret = fscanf(stdin, "%c %ld %ld ", &c, &x, &y);
        }
        i=0;
        index++;
    }
    printf("Index %d\n", index);
    NUMBEROFTEST = index;
}

__global__ void myKernel(inputData* data, outputData* outData){
    int index =  blockIdx.x * blockDim.x + threadIdx.x;
    t_run_test(data[index].realData_1,data[index].imagData_1);

}

void launch(inputData *dataCPU, outputData* outData, char* fileName){
    int threadsPerBlock = 32;
    int blocksPerGrid = NUMBEROFTEST / threadsPerBlock;
    int out_size = NUMBEROFTEST * sizeof(outputData);
    int input_size = NUMBEROFTEST * sizeof(inputData);
    inputData *dataGPUIn;
    outputData* dataGPUOut;
    hipMalloc ((void**)&dataGPUIn, input_size);
    hipMemcpy(dataGPUIn, dataCPU, input_size, hipMemcpyHostToDevice );
    hipMalloc ((void**)&dataGPUOut, out_size);
    hipMemcpy(dataGPUOut, outData, out_size, hipMemcpyHostToDevice );
    myKernel << <blocksPerGrid, threadsPerBlock >> >(dataGPUIn, dataGPUOut);
    hipMemcpy(outData, dataGPUOut, out_size, hipMemcpyDeviceToHost);
    hipFree(dataGPUIn);
    hipFree(dataGPUOut);
}

/***************************************************************************/
int main(int argc, char* argv[] )
{
    inputData* data = (inputData*)malloc(sizeof(inputData) * NUMBEROFTEST);
    outputData* outD = (outputData*)malloc(sizeof(outputData) * NUMBEROFTEST);
    readFile(data);
    launch(data, outD, argv[1]);
    for(int i=0; i<NUMBEROFTEST; i++){
        fprintf(stdout, "ID# %d: ", i);
        for( int i_1 = 0 ; i_1 < NUM_POINTS ; i_1++ )
        {
            fprintf(stdout, " %ld ", data[i].realData_1[i_1] ) ;
        }
        fprintf(stdout, "\n");
    }
    free(data);
    free(outD);
    return 0;
}

