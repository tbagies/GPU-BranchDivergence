#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

int NUMBEROFTEST = 1024;


typedef struct{
    int vertexNum;
    int* adjMatrix;
    int isOriented;
    int hasLoop;
    int vertex;
    int action;
}inputData;

typedef struct{
    int* queue;
    int* wasVisited;
    int* fatherOf;
    double* dist;
}outputData;


void printAdjMatrix(inputData g) {

	int n = g.vertexNum;
	printf("%d\n", n);
	//system(CLEAR_SCREEN);
	for(int i=0; i < n; i++) {
		for(int j=0; j < n; j++)
			printf("%d      ", g.adjMatrix[i*n+j]);
		printf("\n\n");
	}
	//getchar();
} 

__device__ void intializeVector(int* f, int value, int n) {

	for(int i=0; i < n; i++)
		f[i] = value;
}

void printPairs(int* v, int n) {

	printf("  Node |  Father\n\n");
	for(int i=0; i < n; i++)
		printf("  %2d   |    %2d\n", i, v[i]);

	printf("\n* -1 = root\n* -2 = unreachable\n\n");
}

__device__ void breadthFirstSearch(int vertexNum, int* adjMatrix, int vertex, int *wasVisited, int* queue, int* fatherOf) {

	int add = 0;
	int rem = 0;
	int curr; //Current Vertex

	intializeVector(queue, -2, vertexNum);
	intializeVector(fatherOf, -2, vertexNum);

	fatherOf[vertex] = -1;
	queue[add++] = vertex;
	wasVisited[vertex] = 1;
       // printf("rem %d\n", rem);
	while(rem < vertexNum) {
		curr = queue[rem++];
	//	printf("queue[add-1] %d\n ", queue[add-1]);
          //      printf("rem %d \n", rem);
            //    printf("curr %d \n", curr);
	   	if(curr < 0){
	//		printf("break");
			break;
}
		for(int j=0; j < vertexNum; j++) {
//			printf("j %d \n", j);
//			printf("adjMatrix[curr*vertexNum+j] %21f ", adjMatrix[curr*vertexNum+j]);
			if(0 < adjMatrix[curr*vertexNum+j] && adjMatrix[curr*vertexNum+j] < INFINITY)
			{//	printf("if condition");
				if(!wasVisited[j]) {
					queue[add++] = j;
					wasVisited[j] = 1;
			//		printf("queue[add-1] %d \n", queue[add-1]);
					fatherOf[j] = curr;
				}}
		}
	}

	//printPairs(fatherOf, vertexNum);
}

__device__ int allInPartition(int* partition, int n) {

	for(int i=0; i < n; i++)
		if(partition[i] == 0)
			return 0;

	return 1;
}

__device__ int heapIsEmpty(int* heap, int n) {

	for(int i=0; i < n; i++)
		if(heap[i] == 1)
			return 0;

	return 1;
}

__device__ int extractMin(int* heap, double* dist, int n) {

	int minIndex = -1;
	double min = INFINITY;

	for(int i=0; i < n; i++)
		if(heap[i] == 1)
			if(dist[i] < min) {
				min = dist[i];
				minIndex = i;			
			}

	return minIndex;
}

void printPath(int dest, int* father) {

	if(father[dest] == -1)
		printf("%2d ", dest);
	else if(father[dest] == -2)
		printf("UNREACHABLE");
	else {
		printPath(father[dest], father);
		printf("  ->  %2d", dest);
	}
}

__device__ void minimumPathDijkstra(int vertexNum, int* adjMatrix, int vertex, int* heap, double *distance, int* fatherOf) {
       //printf("minimumPathDijkstra");
//	int heap[g->vertexNum];
//	int fatherOf[g->vertexNum];
//	double distance[g->vertexNum];
	int curr;

	intializeVector(heap, 1, vertexNum);
	intializeVector(fatherOf, -2, vertexNum);

	for(int i=0; i < vertexNum; i++)
		distance[i] = INFINITY;

	fatherOf[vertex] = -1;
	distance[vertex] = 0;

	while(!heapIsEmpty(heap, vertexNum)) {
		curr = extractMin(heap, distance, vertexNum);

		if(curr < 0) break;

		heap[curr] = 0;

		for(int j=0; j < vertexNum; j++){
			if(j != curr && 0 < adjMatrix[curr*vertexNum+j])
				if(distance[curr] + adjMatrix[curr*vertexNum+j] < distance[j]) {
					distance[j] = distance[curr] + adjMatrix[curr*vertexNum+j];
					fatherOf[j] = curr;
			}
			// printf("fatherOf[j] distance[j] %d %lf\n", fatherOf[j],distance[j]);
		}
	
	}
	
	/*	for(int i=0; i < vertexNum; i++) {
			printf("d(%2d, %2d) = ", vertex, i);
			printf(" %d = ", distance[i]);
			printPath(i, fatherOf);
			printf("\n");
		}
*/}

__device__ void minimumPathFloydWarshall(int vertexNum, int* adjMatrix, double* dist) {
//	double dist[g->vertexNum][g->vertexNum];

	for(int i=0; i < vertexNum; i++)
		for(int j=0; j < vertexNum; j++) {
			if(i == j)
				dist[i*vertexNum+j] = 0;
			else if(adjMatrix[i*vertexNum+j] == 0)
				dist[i*vertexNum+j] = INFINITY;
			else
				dist[i*vertexNum+j] = adjMatrix[i*vertexNum+j];
		}

	for(int k=0; k < vertexNum; k++)
		for(int i=0; i < vertexNum; i++)
			for(int j=0; j < vertexNum; j++)
				if(dist[i*vertexNum+k] + dist[k*vertexNum+j] < dist[i*vertexNum+j])
					dist[i*vertexNum+j] = dist[i*vertexNum+k] + dist[k*vertexNum+j];
}

__device__ void minimumSpanningTree(int vertexNum, int* adjMatrix, int vertex, int* inPartition) {

//	int* inPartition = calloc(g->vertexNum, sizeof(int));
	int minRow, minCol;
	double min;
	int minSize = 0;
	
	inPartition[vertex] = 1;

	while(!allInPartition(inPartition, vertexNum)) {
		min = INFINITY;
		minRow = -1;
		minCol = -1;
		
		for(int i=0; i < vertexNum; i++)
			if(inPartition[i])
				for(int j=0; j < vertexNum; j++)
					if(!inPartition[j])
						if(0 < adjMatrix[i*vertexNum+j] && adjMatrix[i*vertexNum+j] < min) {
							min = adjMatrix[i*vertexNum+j];
							minRow = i;
							minCol = j;
						}		

		if(minCol < 0) {
			printf("The graph has at least one node that is disconnected\n");
			break;
		}else {
			inPartition[minCol] = 1;
			printf("e(%2d, %2d) = %d\n", minRow, minCol, adjMatrix[minRow*vertexNum+minCol]);
			minSize += adjMatrix[minRow*vertexNum+minCol];
			if(allInPartition(inPartition, vertexNum))
				printf("Minimum Spanning Tree value = %d\n", minSize);
		}
	}
}

//************************************** READING INPUT FROM FILE **************************************** /
void readingInput(inputData *dataCPU, outputData* dataOut){
    int index =0;
    char action;
    int n, isOriented, vertex, i, j;
    int ret = fscanf(stdin, "%c %d %d %d ",&action,&n, &isOriented, &vertex);
    while(ret != EOF){
        dataCPU[index].action = action;
        dataCPU[index].vertexNum = n;
        dataCPU[index].adjMatrix =  (int*)malloc(n*n*sizeof(int));
        dataCPU[index].isOriented = isOriented;
        dataCPU[index].vertex = 0;// vertex;
        dataOut[index].wasVisited = (int*) malloc(n* sizeof(int));
        dataOut[index].queue = (int*) malloc(n* sizeof(int));
        switch(action){
		case 'i':
			dataOut[index].dist = (double*) malloc(n* sizeof(double));
			break;
		case 'f':
			dataOut[index].dist = (double*) malloc(n*n* sizeof(double));
			break;
	}
        dataOut[index].fatherOf = (int*) malloc(n* sizeof(int));
//	printf("%c %d %d %d \n",action,n, isOriented, vertex);
        for(i=0; i < n; i++){
            for(j = 0; j < n; j++) {
                    fscanf(stdin,"%d ", &dataCPU[index].adjMatrix[i*n+j]);
	//	    printf("%d ", dataCPU[index].adjMatrix[i*n+j]);
            }
	//	printf("\n");
	}
        index++;
        ret = fscanf(stdin, "%c %d %d %d ",&action,&n, &isOriented, &vertex);
    }
    NUMBEROFTEST = index;
}

__global__ void myKernel(inputData *dataCPU, outputData* dataOut)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    switch (dataCPU[index].action) {
        case 'b':
            breadthFirstSearch(dataCPU[index].vertexNum,dataCPU[index].adjMatrix , dataCPU[index].vertex, dataOut[index].wasVisited, dataOut[index].queue, dataOut[index].fatherOf);
            break;
        case 'm':
            minimumSpanningTree(dataCPU[index].vertexNum,dataCPU[index].adjMatrix , dataCPU[index].vertex, dataOut[index].queue);
            break;
        case 'i':
            minimumPathDijkstra(dataCPU[index].vertexNum,dataCPU[index].adjMatrix , dataCPU[index].vertex, dataOut[index].wasVisited, dataOut[index].dist, dataOut[index].fatherOf);//, dataCPU.wasVisited);
            break;
       case 'f':
            minimumPathFloydWarshall(dataCPU[index].vertexNum,dataCPU[index].adjMatrix,dataOut[index].dist);
            break;
        default:
            break;
   }
}
//************************************* Writing time in a FILE ******************************************
void writeFile(double time, char* fileName){
    FILE *fp1;
    fp1 = fopen("NUMCudaResults.txt", "a");
    if (fp1 == NULL){
    printf("Problem in a file");
    return;
    }
    fprintf(fp1, "%s ", fileName);
    fprintf(fp1, "%f seconds\n", time);
    fclose(fp1);
}
void launch(inputData *dataCPU, outputData* dataOut, char* fileName){

    int threadsPerBlock = 32;
    int blocksPerGrid = (NUMBEROFTEST) / threadsPerBlock;


    int output_size = (NUMBEROFTEST) * sizeof(outputData);
    int input_size = (NUMBEROFTEST) * sizeof(inputData);
    inputData *dataGPUIn, *in_data = (inputData*)malloc(sizeof(inputData)* NUMBEROFTEST);
    outputData *dataGPUOut, *out_data = (outputData*)malloc(sizeof(outputData)* NUMBEROFTEST);

    int i =0;
    hipEvent_t start, stop; // To satart and stop cluck^M
        float time;
     hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start, 0);
    for(; i<NUMBEROFTEST;i++){
        int* d_adjMatrix;
        int* d_wasVisited;
        int* d_queue;
        int* d_fatherOf;
        double* d_dist;
        hipMalloc((void**)&d_adjMatrix, dataCPU[i].vertexNum * dataCPU[i].vertexNum * sizeof(int));
        hipMemcpy(d_adjMatrix,dataCPU[i].adjMatrix, dataCPU[i].vertexNum * dataCPU[i].vertexNum * sizeof(int),hipMemcpyHostToDevice);
	switch (dataCPU[i].action) {
		case 'm':
			hipMalloc((void**)&d_queue, dataCPU[i].vertexNum * sizeof(int));
                        hipMemcpy(d_queue, dataOut[i].queue, dataCPU[i].vertexNum * sizeof(int),hipMemcpyHostToDevice);
			out_data[i].queue = d_queue;
			break;
	        case 'b':
			hipMalloc((void**)&d_wasVisited, dataCPU[i].vertexNum * sizeof(int));
                        hipMemcpy(d_wasVisited, dataOut[i].wasVisited, dataCPU[i].vertexNum * sizeof(int),hipMemcpyHostToDevice);
                        hipMalloc((void**)&d_queue, dataCPU[i].vertexNum * sizeof(int));
                        hipMemcpy(d_queue, dataOut[i].queue, dataCPU[i].vertexNum * sizeof(int),hipMemcpyHostToDevice);
                        hipMalloc((void**)&d_fatherOf, dataCPU[i].vertexNum * sizeof(int));
                        hipMemcpy(d_fatherOf, dataOut[i].fatherOf, dataCPU[i].vertexNum * sizeof(int),hipMemcpyHostToDevice);
                        out_data[i].fatherOf = d_fatherOf;
                        out_data[i].queue = d_queue;
                        out_data[i].wasVisited = d_wasVisited;
			break;
                case 'i':
			hipMalloc((void**)&d_wasVisited, dataCPU[i].vertexNum * sizeof(int));
       			hipMemcpy(d_wasVisited, dataOut[i].wasVisited, dataCPU[i].vertexNum * sizeof(int),hipMemcpyHostToDevice);
        		hipMalloc((void**)&d_dist, dataCPU[i].vertexNum * sizeof(double));
        		hipMemcpy(d_dist, dataOut[i].dist, dataCPU[i].vertexNum * sizeof(double),hipMemcpyHostToDevice);
        		hipMalloc((void**)&d_fatherOf, dataCPU[i].vertexNum * sizeof(int));
        		hipMemcpy(d_fatherOf, dataOut[i].fatherOf, dataCPU[i].vertexNum * sizeof(int),hipMemcpyHostToDevice);
			out_data[i].fatherOf = d_fatherOf;
			out_data[i].dist = d_dist;
        		out_data[i].wasVisited = d_wasVisited;
	                break;
                case 'f':
			hipMalloc((void**)&d_dist, dataCPU[i].vertexNum * dataCPU[i].vertexNum * sizeof(double));
        		hipMemcpy(d_dist,dataOut[i].dist, dataCPU[i].vertexNum * dataCPU[i].vertexNum * sizeof(double),hipMemcpyHostToDevice);
        		out_data[i].dist = d_dist;            
		   	break;
                default:
                        break;
	}
	in_data[i].vertex = dataCPU[i].vertex;
        in_data[i].vertexNum = dataCPU[i].vertexNum;
        in_data[i].adjMatrix = d_adjMatrix;
        in_data[i].action = dataCPU[i].action;
    }
    hipMalloc ( (void**) &dataGPUIn, input_size);
    hipMemcpy(dataGPUIn, in_data, input_size, hipMemcpyHostToDevice );
    hipMalloc ( (void**) &dataGPUOut, output_size);
    hipMemcpy(dataGPUOut, out_data, output_size, hipMemcpyHostToDevice );
    myKernel << <blocksPerGrid, threadsPerBlock >> >(dataGPUIn, dataGPUOut); // excute on kernel
    hipMemcpy(out_data, dataGPUOut, output_size, hipMemcpyDeviceToHost);
    i=0;
    for(; i<NUMBEROFTEST;i++){
	 switch (dataCPU[i].action) {
        	case 'b':
			hipMemcpy(dataOut[i].fatherOf, out_data[i].fatherOf, dataCPU[i].vertexNum * sizeof(int), hipMemcpyDeviceToHost);
            		break;
        	case 'i':
			hipMemcpy(dataOut[i].fatherOf, out_data[i].fatherOf, dataCPU[i].vertexNum * sizeof(int), hipMemcpyDeviceToHost);
			hipMemcpy(dataOut[i].dist, out_data[i].dist, dataCPU[i].vertexNum * sizeof(double), hipMemcpyDeviceToHost);
            		break;
       		case 'f':
			hipMemcpy(dataOut[i].dist, out_data[i].dist, dataCPU[i].vertexNum *  dataCPU[i].vertexNum *sizeof(double), hipMemcpyDeviceToHost);
            		break;
        	default:
            		break;
    	}
    }
    hipFree(dataGPUIn);
    hipFree(dataGPUOut);
    hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&time, start, stop);
        hipEventDestroy(start);
        hipEventDestroy(stop);
    time = time/1000;
    writeFile(time,fileName);
}
int main(int argc, char* argv[]) {
    inputData *dataCPU = (inputData*)malloc(sizeof(inputData)* NUMBEROFTEST);
    outputData *dataOut = (outputData*)malloc(sizeof(outputData)* NUMBEROFTEST);
    readingInput(dataCPU,dataOut);
    launch(dataCPU,dataOut,argv[1]);
    int index=0;
    for(;index< NUMBEROFTEST; index++) {
        printf("\nID# %d ", index);
	switch (dataCPU[index].action) {
        case 'b':
		printPairs(dataOut[index].fatherOf,dataCPU[index].vertexNum);
        	printf("\n");
    //    	printAdjMatrix(dataCPU[index]);
      //  	printf("\n");
            	break;
       case 'm':  
	//	printf("MST \n");
		break;
       case 'i':
        	for(int i=0; i < dataCPU[index].vertexNum; i++) {
			printf("d(%2d, %2d) ", dataCPU[index].vertex, i);
                        printf(" %6.2lf = ", dataOut[index].dist[i]);
                        printPath(i, dataOut[index].fatherOf);
                        printf(" ");
                }
		break;
       case 'f':	
		for(int i=0; i < dataCPU[index].vertexNum; i++) {
   	             	printf("d(%2d, %2d) = ", dataCPU[index].vertex, i);
			printf("%6.2lf ", dataOut[index].dist[dataCPU[index].vertex*dataCPU[index].vertexNum+i]);
        	}
		break;
	}
	printf("\n");
    }
    return 0;
}






