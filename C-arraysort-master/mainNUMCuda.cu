#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

int NUMBEROFTEST = 1024;

typedef struct{
    int length;
    char typeOfSort;
}inputData;

typedef struct{
    int *pArrayToSort;
}outputData;

__device__ void bubble_sort(int *pArray, int howMany);
__device__ void quick_sort(int * pArray, int start, int end);
__device__ void merge_sort(int* pArray, int length);
__device__ void merge(int * pArray, int * pTempArray, int leftStart, int* mid, int rightEnd);
__device__ void swap(int *x, int *y);
__device__ void gnome_sort(int *arr, int n);
__device__ void comb_sort(int *arr, int n);
__device__ void cocktail_sort(int *arr, int n);
__device__ void insertion_sort(int *arr, int n);
__device__ void odd_even_sort(int *arr, int n);
__device__ void shell_sort(int *arr, int n);
__device__ void selection_sort(int *arr, int n);
__device__ void radix_sort(int* arr, int n);
__device__ int find_largest_number(int arr[], int n);
__device__ int pancake_sort(int* arr, int n);
__device__ void flip(int *arr, int i) ;
__device__ int find_max(int *arr, int n);
__device__ void heap_sort(int *a, int n);
__device__ void build_heap(int *a, int n);
__device__ void satisfy_heap(int *a, int i, int heap_size);

//************************************** READING INPUT FROM FILE **************************************** /
void readingInput(inputData *dataCPU, outputData* dataOut){
    int index =0;
    int length;
    char type;
    int ret = fscanf(stdin, "%d %c",&length,&type);
    while(ret != EOF){
        dataCPU[index].length = length;
        dataCPU[index].typeOfSort = type;
        dataOut[index].pArrayToSort = (int*)malloc(length * sizeof(int));
        int i;
        for(i=0; i<length; i++)
            fscanf(stdin, "%d",  &dataOut[index].pArrayToSort[i]);
        index++;
        ret = fscanf(stdin, "%d %c",&length,&type);
    }
    NUMBEROFTEST = index;
}

__global__ void myKernel(inputData *dataCPU, outputData* dataOut)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    switch(dataCPU[index].typeOfSort){
        case 'b' :
            printf("Using bubblesort\n");
            bubble_sort(dataOut[index].pArrayToSort, dataCPU[index].length);
            break;
        case 'q' :
            printf("Using quicksort\n");
            quick_sort(dataOut[index].pArrayToSort, 0, dataCPU[index].length - 1);
            break;
        case 'm' :
            printf("Using mergesort\n");
            merge_sort(dataOut[index].pArrayToSort, dataCPU[index].length);
            break;
        case 'o' :
            printf("Using odd_even_sort\n");
            odd_even_sort(dataOut[index].pArrayToSort, dataCPU[index].length);
            break;

        case 'c' :
            printf("Using cocktail_sort\n");
            cocktail_sort(dataOut[index].pArrayToSort, dataCPU[index].length);
            break;
        case 't' :
            printf("Using comb_sort\n");
            comb_sort(dataOut[index].pArrayToSort, dataCPU[index].length);
            break;
        case 'g' :
            printf("Using gnome_sort\n");
            gnome_sort(dataOut[index].pArrayToSort, dataCPU[index].length);
            break;
	case 'i' :
            printf("Using insertion_sort\n");
            insertion_sort(dataOut[index].pArrayToSort, dataCPU[index].length);
            break;
       case 'h' :
            printf("Using shell_sort\n");
            shell_sort(dataOut[index].pArrayToSort, dataCPU[index].length);
            break;
	case 's' :
            printf("Using selection_sort\n");
            selection_sort(dataOut[index].pArrayToSort, dataCPU[index].length);
            break;
	case 'r' :
            printf("Using radix_sort\n");
            radix_sort(dataOut[index].pArrayToSort, dataCPU[index].length);
            break;
	case 'p' :
            printf("Using pancake_sort\n");
            pancake_sort(dataOut[index].pArrayToSort, dataCPU[index].length);
            break;
	case 'u' :
            printf("Using heap_sort\n");
            heap_sort(dataOut[index].pArrayToSort, dataCPU[index].length);
            break;
        default:
            printf("No Correct Sort Chosen\n");
            break;
    }

}
//************************************* Writing time in a FILE ******************************************
void writeFile(double time, char* fileName){
    FILE *fp1;
    fp1 = fopen("NUMCudaResults.txt", "a");
    if (fp1 == NULL){
        printf("Problem in a file");
        return;
    }
    fprintf(fp1, "%s ", fileName);
    fprintf(fp1, "%f seconds\n", time);
    fclose(fp1);
}
void launch(inputData *dataCPU, outputData* dataOut, char* fileName){

    int threadsPerBlock = 32;
    int blocksPerGrid = (NUMBEROFTEST) / threadsPerBlock;
    int output_size = (NUMBEROFTEST) * sizeof(outputData);
    int input_size = (NUMBEROFTEST) * sizeof(inputData);
    inputData *dataGPUIn, *in_data = (inputData*)malloc(sizeof(inputData)* NUMBEROFTEST);
    outputData *dataGPUOut, *out_data = (outputData*)malloc(sizeof(outputData)* NUMBEROFTEST);
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    int i =0;
    hipEventRecord(start);
    for(; i<NUMBEROFTEST;i++){
        int* d_array;
        hipMalloc((void**)&d_array, dataCPU[i].length*sizeof(int));
        hipMemcpy(d_array,dataOut[i].pArrayToSort, dataCPU[i].length*sizeof(int),hipMemcpyHostToDevice);
        in_data[i].length = dataCPU[i].length;
        in_data[i].typeOfSort = dataCPU[i].typeOfSort;
        out_data[i].pArrayToSort = d_array;
    }
    hipMalloc ( (void**) &dataGPUIn, input_size);
    hipMemcpy(dataGPUIn, in_data, input_size, hipMemcpyHostToDevice );
    hipMalloc ( (void**) &dataGPUOut, output_size);
    hipMemcpy(dataGPUOut, out_data, output_size, hipMemcpyHostToDevice );
    myKernel << <blocksPerGrid, threadsPerBlock >> >(dataGPUIn, dataGPUOut); // excute on kernel
    hipMemcpy(out_data, dataGPUOut, output_size, hipMemcpyDeviceToHost);
    i=0;
    for(; i<NUMBEROFTEST;i++){
        hipMemcpy(dataOut[i].pArrayToSort, out_data[i].pArrayToSort, dataCPU[i].length*sizeof(int), hipMemcpyDeviceToHost);
    }
    hipFree(dataGPUIn);
    hipFree(dataGPUOut);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    writeFile((milliseconds/1000),fileName);
}
int main (int args, char* argv[])
{
    inputData *dataCPU = (inputData*)malloc(sizeof(inputData)* NUMBEROFTEST);
    outputData *dataOut = (outputData*)malloc(sizeof(outputData)* NUMBEROFTEST);
    readingInput(dataCPU,dataOut);
    launch(dataCPU,dataOut,argv[1]);
    int index=0,i;
    for(;index< NUMBEROFTEST; index++) {
        printf("ID# %d ", index, dataCPU[index].length);

        for(i = 0 ; i < dataCPU[index].length ; i++){
            printf("%d ", *(dataOut[index].pArrayToSort + i));
        }
        printf("\n");
    }

    return 0;
}

/********* https://github.com/eddyerburgh/c-sorting-algorithms******/

__device__ void quick_sort(int arr[], int first_index, int last_index) {
  // declaring index variables
  int pivotIndex, temp, index_a, index_b;

  if (first_index < last_index) {
    // assigning first element index as pivot element
    pivotIndex = first_index;
    index_a = first_index;
    index_b = last_index;

    // Sorting in Ascending order with quick sort
    while (index_a < index_b) {
      while (arr[index_a] <= arr[pivotIndex] && index_a < last_index) {
        index_a++;
      }
      while (arr[index_b] > arr[pivotIndex]) {
        index_b--;
      }

      if (index_a < index_b) {
        // Swapping operation
        temp = arr[index_a];
        arr[index_a] = arr[index_b];
        arr[index_b] = temp;
      }
    }

    // At the end of first iteration, swap pivot element with index_b element
    temp = arr[pivotIndex];
    arr[pivotIndex] = arr[index_b];
    arr[index_b] = temp;

    // Recursive call for quick sort, with partitioning
    quick_sort(arr, first_index, index_b - 1);
    quick_sort(arr, index_b + 1, last_index);
  }
}

__device__ void merge(
    int* arr, int* left_arr, int left_count, int* right_arr, int right_count) {
  int i, j, k;

  i = 0;
  j = 0;
  k = 0;

  while (i < left_count && j < right_count) {
    if (left_arr[i] < right_arr[j])
      arr[k++] = left_arr[i++];
    else
      arr[k++] = right_arr[j++];
  }
  while (i < left_count) arr[k++] = left_arr[i++];
  while (j < right_count) arr[k++] = right_arr[j++];
}

__device__ void merge_sort(int arr[], int n) {
  int mid, i, *L, *right_arr;
  if (n < 2) return;
  mid = n / 2;

  L = (int*)malloc(mid * sizeof(int));
  right_arr = (int*)malloc((n - mid) * sizeof(int));

  for (i = 0; i < mid; i++) L[i] = arr[i];
  for (i = mid; i < n; i++) right_arr[i - mid] = arr[i];

  merge_sort(L, mid);
  merge_sort(right_arr, n - mid);
  merge(arr, L, mid, right_arr, n - mid);
  free(L);
  free(right_arr);
}
__device__ void bubble_sort(int arr[], int n) {
  int temp;
  int i;
  int j;
  
  i = 0;
  while (i < n) {
    j = 0;
    while (j < i) {
      if (arr[j] > arr[i]) {
        temp = arr[j];
        arr[j] = arr[i];
        arr[i] = temp;
      }
      j++;
    }
    i++;
  }
}
__device__ void odd_even_sort(int *arr, int n) {
  bool sorted = false;  // Initially array is unsorted
  int temp;

  while (!sorted) {
    sorted = true;

    // Perform Bubble sort on odd indexed element
    for (int i = 1; i <= n - 2; i += 2) {
      if (arr[i] > arr[i + 1]) {
        temp = arr[i + 1];
        arr[i + 1] = arr[i];
        arr[i] = temp;
        sorted = false;
      }
    }

    // Perform Bubble sort on even indexed element
    for (int i = 0; i <= n - 2; i += 2) {
      if (arr[i] > arr[i + 1]) {
        temp = arr[i + 1];
        arr[i + 1] = arr[i];
        arr[i] = temp;
        sorted = false;
      }
    }
  }
}
__device__ void cocktail_sort(int *arr, int n) {
    bool swapped = true;
    int i = 0;
    int j = n - 1;

    while (i < j && swapped) {
        swapped = false;
        for (int k = i; k < j; k++) {
            if (arr[k] > arr[k + 1]) {
            int temp = arr[k];
            arr[k] = arr[k + 1];
            arr[k + 1] = temp;
            swapped = true;
            }
        }
        j--;
        if (swapped) {
            swapped = false;
            for (int k = j; k > i; k--) {
                if (arr[k] < arr[k - 1]) {
                    int temp = arr[k];
                    arr[k] = arr[k - 1];
                    arr[k - 1] = temp;
                    swapped = true;
                }
            }
        }
        i++;
    }
}

__device__ void comb_sort(int *arr, int n) {
    int temp;
    int i;
    int j;
    int gap = n;
    int shrink = 3;
    bool swapped;

    while (gap != 1 || swapped == true) {
        gap = (gap / shrink);
        if (gap < 1) {
            gap = 1;
        }
        swapped = false;
        for (int i = 0; i < n - gap; i++) {
            if (arr[i] > arr[i + gap]) {
            temp = arr[i + gap];
            arr[i + gap] = arr[i];
            arr[i] = temp;
            swapped = true;
            }
        }
    }
}

__device__ void gnome_sort(int *arr, int n) {
    int i = 0;
    int temp;

    while (i < n) {
        if (i == 0 || arr[i] >= arr[i - 1]) {
            i++;
        } else {
            temp = arr[i];
            arr[i] = arr[i - 1];
            arr[i - 1] = temp;
            i--;
        }
    }
}
__device__ void insertion_sort(int *arr, int n) {
  for (int i = 0; i < n; i++) {
    int j = i;
    while (j > 0 && arr[j - 1] > arr[j]) {
      int temp = arr[j];
      arr[j] = arr[j - 1];
      arr[j - 1] = temp;
      j--;
    }
  }
}

__device__ void shell_sort(int *arr, int n) {
  int j;

  for (int gap = n / 2; gap > 0; gap /= 2) {
    for (int i = gap; i < n; ++i) {
      int temp = arr[i];
      for (j = i; j >= gap && temp < arr[j - gap]; j -= gap) {
        arr[j] = arr[j - gap];
      }
      arr[j] = temp;
    }
  }
}

__device__ void selection_sort(int *arr, int n) {
  int i, j;

  for (j = 0; j < n - 1; j++) {
    int iMin = j;

    for (i = j + 1; i < n; i++) {
      if (arr[i] < arr[iMin]) {
        iMin = i;
      }
    }

    if (iMin != j) {
      int temp = arr[j];
      arr[j] = arr[iMin];
      arr[iMin] = temp;
    }
  }
}

__device__ int find_largest_number(int *arr, int n) {

  int i;
  int largest_number = -1;

  for (i = 0; i < n; i++) {
    if (arr[i] > largest_number) largest_number = arr[i];
  }

  return largest_number;
}

__device__ void radix_sort(int* arr, int n) {

  // Base 10 is used
  int i;
  int *semi_sorted = (int*)malloc(n*sizeof(int));
  int significant_digit = 1;
  int largest_number = find_largest_number(arr, n);

  // Loop until we reach the largest significant digit
  while (largest_number / significant_digit > 0) {


    int bucket[10] = {0};

    // Counts the number of "keys" or digits that will go into each bucket
    for (i = 0; i < n; i++) bucket[(arr[i] / significant_digit) % 10]++;

    /**
     * Add the count of the previous buckets,
     * Acquires the indexes after the end of each bucket location in the arr
     * Works similar to the count sort algorithm
     **/
    for (i = 1; i < 10; i++) bucket[i] += bucket[i - 1];

    // Use the bucket to fill a "semi_sorted" arr
    for (i = n - 1; i >= 0; i--)
      semi_sorted[--bucket[(arr[i] / significant_digit) % 10]] = arr[i];


    for (i = 0; i < n; i++) arr[i] = semi_sorted[i];

    // Move to next significant digit
    significant_digit *= 10;
  }
}

__device__ void flip(int *arr, int i) {
  int temp, start = 0;
  while (start < i) {
    temp = arr[start];
    arr[start] = arr[i];
    arr[i] = temp;
    start++;
    i--;
  }
}

__device__ int find_max(int *arr, int n) {
  int mi, i;
  for (mi = 0, i = 0; i < n; ++i)
    if (arr[i] > arr[mi]) mi = i;
  return mi;
}


__device__ int pancake_sort(int* arr, int n) {
  for (int curr_size = n; curr_size > 1; --curr_size) {
    int mi = find_max(arr, curr_size);
    if (mi != curr_size - 1) {
      flip(arr, mi);

      flip(arr, curr_size - 1);
    }
  }
}
__device__ void heap_sort(int a[], int n) {
  build_heap(a, n);
  int heap_size, i, temp;
  heap_size = n - 1;
  for (i = heap_size; i >= 0; i--) {
    temp = a[0];
    a[0] = a[heap_size];
    a[heap_size] = temp;
    heap_size--;
    satisfy_heap(a, 0, heap_size);
  }
}

__device__ void build_heap(int a[], int n) {
  int i, heap_size;
  heap_size = n - 1;
  for (i = (n / 2); i >= 0; i--) {
    satisfy_heap(a, i, heap_size);
  }
}

__device__ void satisfy_heap(int a[], int i, int heap_size) {
  int l, r, largest, temp;
  l = 2 * i;
  r = 2 * i + 1;
  if (l <= heap_size && a[l] > a[i]) {
    largest = l;
  } else {
    largest = i;
  }
  if (r <= heap_size && a[r] > a[largest]) {
    largest = r;
  }
  if (largest != i) {
    temp = a[i];
    a[i] = a[largest];
    a[largest] = temp;
    satisfy_heap(a, largest, heap_size);
  }
}
