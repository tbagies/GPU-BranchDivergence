#include "hip/hip_runtime.h"
//https://github.com/VAR-solutions/Algorithms

#include <stdio.h>
#include <limits.h>
int NUMBEROFTEST = 1024;


typedef struct{
    int n;
    int k;
    int* c;
    int* m;
    int* t;
    bool *s;
    char* x;
    char* y;
    char action;
}inputData;

typedef struct{
    int result;
}outputData;

// ****************************** binomialCoeff ********************************* 

/*__device__ int min(int a, int b)
{
  return (a < b) ? a : b;
}*/
__device__ int binomialCoeff(int n, int k, int* C)
{
  //int C[n + 1][k + 1];
  int i, j;
  for (i = 0; i <= n; i++)
  {
    for (j = 0; j <= min(i, k); j++)
    {
      if (j == 0 || j == i)
        C[i*(k+1)+j] = 1;

      else
        C[i*(k+1)+j] = C[(i - 1)*(k+1)+j - 1] + C[(i - 1)*(k+1)+j];
    }
  }

  return C[n*(k+1)+k];
}


// ****************************** END binomialCoeff ********************************* 

// Matrix Ai has dimension p[i-1] x p[i] for i = 1..n 
 __device__ int MatrixChainOrder(int *p, int n, int* m) 
{ 
  
    /* For simplicity of the program, one extra row and one 
       extra column are allocated in m[][].  0th row and 0th 
       column of m[][] are not used */
   // int m[n][n]; 
    int i, j, k, L, q; 
  
    /* m[i,j] = Minimum number of scalar multiplications needed 
       to compute the matrix A[i]A[i+1]...A[j] = A[i..j] where 
       dimension of A[i] is p[i-1] x p[i] */
  
    // cost is zero when multiplying one matrix. 
    for (i=1; i<n; i++) 
        m[i*n+i] = 0; 
  
    // L is chain length. 
    for (L=2; L<n; L++) 
    { 
        for (i=1; i<n-L+1; i++) 
        { 
            j = i+L-1; 
            m[i*n+j] = INT_MAX; 
            for (k=i; k<=j-1; k++) 
            { 
                // q = cost/scalar multiplications 
                q = m[i*n+k] + m[(k+1)*n+j] + p[i-1]*p[k]*p[j];
                if (q < m[i*n+j]) 
                    m[i*n+j] = q; 
            } 
        } 
    }
    return m[n+n-1]; 
} 
  /******************** END MATRIX MULTPICATION *******************************/

/**************** FIB *****************************************************/
__device__ int fib(int n, int* f) 
{ 
  /* Declare an array to store Fibonacci numbers. */
//  int f[n+2];   // 1 extra to handle case, n = 0 
  int i; 
  
  /* 0th and 1st number of the series are 0 and 1*/
  f[0] = 0; 
  f[1] = 1; 
  
  for (i = 2; i <= n; i++) 
  { 
      /* Add the previous 2 numbers in the series 
         and store it */
      f[i] = f[i-1] + f[i-2]; 
  } 
  
  return f[n]; 
} 
 /******************** END FIB ************************************/

/************************** KADANE *******************************
Kadane's algorithm finds the maximum subarray(contigious subsequence) sum.
    let f(i) be the maximum sum of any non-empty subarray ending at i.
    f(i) = {
                max(arr[i], f[i-1] + arr[i])    otherwise
                arr[0]                            when i = 0
            }
    */
//Time Complexity  = O(n)
//Space Complexity = O(n)
__device__ int max_subarray_sum(int* arr, int size,int* f){
   // int f[size];
    int maximum_sum;

    f[0] = arr[0];
    maximum_sum = arr[0];
    for(int i=1; i< size; i++){
        f[i] = max(arr[i], f[i-1]+ arr[i]);
        //keep track of maximum sum ending at ith position found so far.
        maximum_sum = max(f[i],maximum_sum);
    }
    return maximum_sum;
}
/************************** END KADANE *******************************/

/*************************** KNAPSACK ********************************/
__device__ int knapSack(int W,int* wt,int* v,int n,int* T){

    //int T[n + 1][W + 1];

    for (int i = 0; i <= n; i++)
    {

        for (int j = 0; j <= W; j++)
        {
            if (i == 0 || j == 0)
                T[i*(W+1)+j] = 0;

            else if (wt[i - 1] <= j)
                T[i*(W+1)+j] = max(v[i - 1] + T[(i - 1)*(W+1)+j - wt[i - 1]], T[(i - 1)*(W+1)+j]);

            else
                T[i*W+j] = T[(i - 1)*(W+1)+j];
        }
    }

    return T[n*(W+1)+W];
}
/************************** END KNAPSACK ********************************/

/********************* SUBSETSUM ********************************/
__device__ bool isSubsetSum(int *set, int n, int sum,bool* subset){
//  bool subset[n+1][sum+1];
  for (int i = 0; i <= n; i++)
  	subset[i*(sum+1)+0] = true;
  for (int i = 1; i <= sum; i++)
  	subset[0*(sum+1)+i] = false;
  for (int i = 1; i <= n; i++){
    for (int j = 1; j <= sum; j++){
      if(j<set[i-1])
      	subset[i*(sum+1)+j] = subset[(i-1)*(sum+1)+j];
      if (j >= set[i-1])
      	subset[i*(sum+1)+j] = subset[(i-1)*(sum+1)+j] || subset[(i - 1)*(sum+1)+j-set[i-1]];
    }
  }
  return subset[n*(sum+1)+sum];
}
/********************* END SUBSETSUM ********************************/
/************* FROM GITHUB SIMILAR TO  https://www.geeksforgeeks.org/longest-common-subsequence-dp-4/ ******/
__device__ int lcs(char* X, char* Y, int m, int n, int* L){
  // int L[m+1][n+1]; 
   int i, j; 
   
   /* Following steps build L[m+1][n+1] in bottom up fashion. Note  
      that L[i][j] contains length of LCS of X[0..i-1] and Y[0..j-1] */
   for (i=0; i<=m; i++) 
   { 
     for (j=0; j<=n; j++) 
     { 
       if (i == 0 || j == 0) 
         L[i*(n+1)+j] = 0; 
   
       else if (X[i-1] == Y[j-1]) 
         L[i*(n+1)+j] = L[(i-1)*(n+1)+j-1] + 1; 
   
       else
         L[i*(n+1)+j] = max(L[(i-1)*(n+1)+j], L[i*(n+1)+j-1]); 
     } 
   } 
     
   /* L[m][n] contains length of LCS for X[0..n-1] and Y[0..m-1] */
   return L[m*(n+1)+n]; 
}
/******************************** END longest-common-subsequence ************/
/*************** https://www.geeksforgeeks.org/edit-distance-dp-5/ **********/
// Utility function to find the minimum of three numbers 
__device__ int min(int x, int y, int z) 
{ 
    return min(min(x, y), z); 
}
__device__ int editDistDP(char* str1, char* str2, int m, int n, int* dp) 
{ 
    // Create a table to store results of subproblems 
  //  int dp[m+1][n+1]; 
  
    // Fill d[][] in bottom up manner 
    for (int i=0; i<=m; i++) 
    { 
        for (int j=0; j<=n; j++) 
        { 
            // If first string is empty, only option is to 
            // insert all characters of second string 
            if (i==0) 
                dp[i*(n+1)+j] = j;  // Min. operations = j 
  
            // If second string is empty, only option is to 
            // remove all characters of second string 
            else if (j==0) 
                dp[i*(n+1)+j] = i; // Min. operations = i 
  
            // If last characters are same, ignore last char 
            // and recur for remaining string 
            else if (str1[i-1] == str2[j-1]) 
                dp[i*(n+1)+j] = dp[(i-1)*(n+1)+j-1]; 
  
            // If the last character is different, consider all 
            // possibilities and find the minimum 
            else
                dp[i*(n+1)+j] = 1 + min(dp[i*(n+1)+j-1],  // Insert 
                                   dp[(i-1)*(n+1)+j],  // Remove 
                                   dp[(i-1)*(n+1)+j-1]); // Replace 
        } 
    } 
  
    return dp[m*(n+1)+n]; 
} 
/***************************** END EDIT DISTANCE ***************/

/****** EGG DROPIING PUZZEL ************************/
__device__ int eggDrop(int n, int k, int* eggFloor) 
{ 
   // int eggFloor[n+1][k+1]; 
    int res; 
    int i, j, x; 
  
    for (i = 1; i <= n; i++) 
    { 
        eggFloor[i*(k+1)+1] = 1; 
        eggFloor[i*(k+1)+0] = 0; 
    } 
  
    for (j = 1; j <= k; j++) 
        eggFloor[1*(k+1)+j] = j; 
  
    for (i = 2; i <= n; i++) 
    { 
        for (j = 2; j <= k; j++) 
        { 
            eggFloor[i*(k+1)+j] = INT_MAX; 
            for (x = 1; x <= j; x++) 
            { 
                res = 1 + max(eggFloor[(i-1)*(k+1)+x-1], eggFloor[i*(k+1)+j-x]); 
                if (res < eggFloor[i*(k+1)+j]) 
                    eggFloor[i*(k+1)+j] = res; 
            } 
        } 
    } 
  
    return eggFloor[n*(k+1)+k]; 
} 

/********************** END EGG FROPPING PUZZEL ***************************/

/************************** STRATGEY GAME *********************************/
// Returns optimal value possible that a player can collect from 
// an array of coins of size n. Note than n must be even 
__device__ int optimalStrategyOfGame(int* arr, int n, int* table) 
{ 
    // Create a table to store solutions of subproblems 
  //  int table[n][n];
	int gap, i, j, x, y, z; 
    // Fill table using above recursive formula. Note that the table 
    // is filled in diagonal fashion (similar to http:// goo.gl/PQqoS), 
    // from diagonal elements to table[0][n-1] which is the result. 
    for (gap = 0; gap < n; ++gap) {
        for (i = 0, j = gap; j < n; ++i, ++j) { 
            // Here x is value of F(i+2, j), y is F(i+1, j-1) and 
            // z is F(i, j-2) in above recursive formula 
            x = ((i + 2) <= j) ? table[(i + 2)*n+j] : 0; 
            y = ((i + 1) <= (j - 1)) ? table[(i + 1)* n + j - 1] : 0; 
            z = (i <= (j - 2)) ? table[i*n + j - 2] : 0; 
 	    table[i*n+j] = max(arr[i] + min(x, y), arr[j] + min(y, z)); 
        } 
    } 
    return table[n - 1]; 
} 
/******************** END STRATGEY GAME *********************************/

/************** Longest Increasing Subsequence ********
* The program finds the length of the Longest subsequence (may not be
 * continuous) such that the subsequence is in increasing order
***************/
__device__ int findMax(int *arr, int length) {
    int max = 0;
    for (int i = 0; i < length; i++) {
        if (max < arr[i]) {
            max = arr[i];	
        }			
    }
    return max;
}

__device__ int findLongestIncSubLength(int *arr,int length, int* dp) {
    //int dp[length];
    int i, j, max = 0;
	
    // Initialize Longest Increasing Subsequence values
    for (i = 0; i < length; i++) {
        dp[i] = 1;			
    }
    
    for (i = 1; i < length; i++) {
        for (j = 0; j < i; j++) {
            if (arr[i] > arr[j] && dp[i] < dp[j] + 1) {
                dp[i] = dp[j] + 1;	
            }				
        }
    }
    max = findMax(dp, length);
    return max;
}
/***************************** END Longest Increasing Subsequence ******************************************/

void readingInput(inputData *dataCPU, outputData* dataOut){
    int index =0;
    char action;
    int n, k, i;
    int ret = fscanf(stdin, "%c ",&action);
    while(ret != EOF){
	switch(action){
		case 'c':
			ret = fscanf(stdin, "%d %d ",&n, &k);
			dataCPU[index].action = action;
        		dataCPU[index].n = n;
        		dataCPU[index].k = k;
        		dataCPU[index].c = (int*)malloc(sizeof(int)*(n+1)*(k+1));
			break;
		case 'm':
                        ret = fscanf(stdin, "%d ",&n);
			dataCPU[index].action = action;
        		dataCPU[index].n = n;
        		dataCPU[index].c = (int*)malloc(sizeof(int)*n);
			for(i=0;i<n;i++)
				ret = fscanf(stdin, "%d ",&dataCPU[index].c[i]);
			dataCPU[index].m = (int*)malloc(sizeof(int)*n * n);
			break;
		case 'f':
                        ret = fscanf(stdin, "%d ",&n);
                        dataCPU[index].action = action;
                        dataCPU[index].n = n;
                        dataCPU[index].c = (int*)malloc(sizeof(int)*(n+2));
			break;
		case 'k':
                        ret = fscanf(stdin, "%d ",&n);
                        dataCPU[index].action = action;
                        dataCPU[index].n = n;
                        dataCPU[index].c = (int*)malloc(sizeof(int)*n);
			for(i=0;i<n;i++)
                                ret = fscanf(stdin, "%d ",&dataCPU[index].c[i]);
			dataCPU[index].m = (int*)malloc(sizeof(int)*n);
                        break;
		case 'n':
			ret = fscanf(stdin, "%d %d ",&n, &k);
                        dataCPU[index].action = action;
                        dataCPU[index].n = n;
			dataCPU[index].k=k;
                        dataCPU[index].c = (int*)malloc(sizeof(int)*n);
                        for(i=0;i<n;i++)
                                ret = fscanf(stdin, "%d ",&dataCPU[index].c[i]);
                        dataCPU[index].m = (int*)malloc(sizeof(int)*n);
			for(i=0;i<n;i++)
                                ret = fscanf(stdin, "%d ",&dataCPU[index].m[i]);
			dataCPU[index].t = (int*)malloc(sizeof(int)*(n+1)*(k+1));
                        break;
		case 's':
                        ret = fscanf(stdin, "%d %d ",&n, &k);
                        dataCPU[index].action = action;
                        dataCPU[index].n = n;
                        dataCPU[index].k=k;
                        dataCPU[index].c = (int*)malloc(sizeof(int)*n);
                        for(i=0;i<n;i++)
                                ret = fscanf(stdin, "%d ",&dataCPU[index].c[i]);
                        dataCPU[index].s = (bool*)malloc(sizeof(bool)*(n+1)*(k+1));
                        break;
		case 'l':
			ret = fscanf(stdin, "%d %d ",&n, &k);
                        dataCPU[index].action = action;
                        dataCPU[index].n = n;
                        dataCPU[index].k=k;
                        dataCPU[index].x = (char*)malloc(sizeof(char)*n);
			for(i=0;i<n;i++){
                                ret = fscanf(stdin, "%c ",&dataCPU[index].x[i]);//dataCPU[index].x[i]);
			}
			dataCPU[index].y = (char*)malloc(sizeof(char)*k);
                        for(i=0;i<k;i++)
                                ret = fscanf(stdin, "%c ",&dataCPU[index].y[i]);
			 dataCPU[index].m = (int*)malloc(sizeof(int)*(n+1)*(k+1));
                    
			break;
		case 'e':
                        ret = fscanf(stdin, "%d %d ",&n, &k);
                        dataCPU[index].action = action;
                        dataCPU[index].n = n;
                        dataCPU[index].k=k;
                        dataCPU[index].x = (char*)malloc(sizeof(char)*n);
                        for(i=0;i<n;i++)
                                ret = fscanf(stdin, "%c ",&dataCPU[index].x[i]);
                        dataCPU[index].y = (char*)malloc(sizeof(char)*k);
                        for(i=0;i<k;i++)
                                ret = fscanf(stdin, "%c ",&dataCPU[index].y[i]);
                         dataCPU[index].m = (int*)malloc(sizeof(int)*(n+1)*(k+1));
                        break;
		case 'g':
			ret = fscanf(stdin, "%d %d ",&n, &k);
                        dataCPU[index].action = action;
                        dataCPU[index].n = n;
                        dataCPU[index].k=k;
                        dataCPU[index].c = (int*)malloc(sizeof(int)*(n+1)*(k+1));
			break;
		case 'o':
			ret = fscanf(stdin, "%d ",&n);
                        dataCPU[index].action = action;
                        dataCPU[index].n = n;
                        dataCPU[index].c = (int*)malloc(sizeof(int)*n);
                        for(i=0;i<n;i++)
                                ret = fscanf(stdin, "%d ",&dataCPU[index].c[i]);
                        dataCPU[index].m = (int*)malloc(sizeof(int)*n*n);
			break;
		case 'i':
			ret = fscanf(stdin, "%d ",&n);
                        dataCPU[index].action = action;
                        dataCPU[index].n = n;
                        dataCPU[index].c = (int*)malloc(sizeof(int)*n);
                        for(i=0;i<n;i++)
                                ret = fscanf(stdin, "%d ",&dataCPU[index].c[i]);
                        dataCPU[index].m = (int*)malloc(sizeof(int)*n);
			break;
		default:
			printf("DEFAULT ID %d\n", index);
			break;
        }
	index++;
    	ret = fscanf(stdin, "\n%c ",&action);
    }
    NUMBEROFTEST = index;
}
__global__ void myKernel(inputData *dataCPU, outputData* dataOut)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    switch (dataCPU[index].action) {
        case 'c':
         dataOut[index].result=binomialCoeff(dataCPU[index].n,dataCPU[index].k, dataCPU[index].c);
	break;
	case 'm':
		dataOut[index].result= MatrixChainOrder(dataCPU[index].c,dataCPU[index].n,dataCPU[index].m);
		break;
	case 'f':
		dataOut[index].result=fib(dataCPU[index].n,dataCPU[index].c);
		break;
	case 'k':
		dataOut[index].result=max_subarray_sum(dataCPU[index].c,dataCPU[index].n,dataCPU[index].m);
        	break;
	case 'n':
		dataOut[index].result=knapSack(dataCPU[index].k,dataCPU[index].m,dataCPU[index].c,dataCPU[index].n,dataCPU[index].t);
		break;
        case 's':
		dataOut[index].result=isSubsetSum(dataCPU[index].c, dataCPU[index].n, dataCPU[index].k,dataCPU[index].s);
		break;
	case 'l':
		dataOut[index].result = lcs( dataCPU[index].x, dataCPU[index].y, dataCPU[index].n, dataCPU[index].k,dataCPU[index].m);
		break;
	case 'e':
		dataOut[index].result = editDistDP( dataCPU[index].x, dataCPU[index].y, dataCPU[index].n, dataCPU[index].k,dataCPU[index].m);
		break;
	case 'g':
		dataOut[index].result = eggDrop(dataCPU[index].n,dataCPU[index].k, dataCPU[index].c);
		break;
	case 'o':
		dataOut[index].result = optimalStrategyOfGame(dataCPU[index].c, dataCPU[index].n, dataCPU[index].m);
		break;
	case 'i':
		 dataOut[index].result = findLongestIncSubLength(dataCPU[index].c, dataCPU[index].n, dataCPU[index].m);
		break;
	default:
            break;
   }
}
//************************************* Writing time in a FILE ******************************************
void writeFile(double time, char* fileName){
    FILE *fp1;
    fp1 = fopen("NUMCudaResults.txt", "a");
    if (fp1 == NULL){
    printf("Problem in a file");
    return;
    }
    fprintf(fp1, "%s ", fileName);
    fprintf(fp1, "%f seconds\n", time);
    fclose(fp1);
}
void launch(inputData *dataCPU, outputData* dataOut, char* fileName){

    int threadsPerBlock = 32;
    int blocksPerGrid = (NUMBEROFTEST) / threadsPerBlock;


    int output_size = (NUMBEROFTEST) * sizeof(outputData);
    int input_size = (NUMBEROFTEST) * sizeof(inputData);
    inputData *dataGPUIn, *in_data = (inputData*)malloc(sizeof(inputData)* NUMBEROFTEST);
    outputData *dataGPUOut, *out_data = (outputData*)malloc(sizeof(outputData)* NUMBEROFTEST);

    int i =0;
    hipEvent_t start, stop; // To satart and stop cluck^M
        float time;
     hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start, 0);
    for(; i<NUMBEROFTEST;i++){
        int* d_c;
	int* d_m;
        int* d_t;
	bool* d_s;
	char* d_x;
	char* d_y;
	switch (dataCPU[i].action) {
		case 'c':
		case 'g':
			hipMalloc((void**)&d_c, (dataCPU[i].n+1) * (dataCPU[i].k+1) * sizeof(int));
        		hipMemcpy(d_c,dataCPU[i].c,(dataCPU[i].n+1) * (dataCPU[i].k+1) * sizeof(int) ,hipMemcpyHostToDevice);
			break;
	        case 'm':
		case 'o':
			hipMalloc((void**)&d_c, dataCPU[i].n * sizeof(int));
                        hipMemcpy(d_c,dataCPU[i].c, dataCPU[i].n * sizeof(int) ,hipMemcpyHostToDevice);
			hipMalloc((void**)&d_m, dataCPU[i].n * dataCPU[i].n * sizeof(int));
                        hipMemcpy(d_m,dataCPU[i].m, dataCPU[i].n * dataCPU[i].n * sizeof(int) ,hipMemcpyHostToDevice);
			in_data[i].m = d_m;
			break;
                case 'f':
			hipMalloc((void**)&d_c, (2+dataCPU[i].n) * sizeof(int));
                        hipMemcpy(d_c,dataCPU[i].c, (dataCPU[i].n+2) * sizeof(int) ,hipMemcpyHostToDevice);
	                break;
                case 'k':
		case 'i':
			hipMalloc((void**)&d_c, dataCPU[i].n * sizeof(int));
                        hipMemcpy(d_c,dataCPU[i].c, dataCPU[i].n * sizeof(int) ,hipMemcpyHostToDevice);
                        hipMalloc((void**)&d_m, dataCPU[i].n * sizeof(int));
                        hipMemcpy(d_m,dataCPU[i].m, dataCPU[i].n * sizeof(int) ,hipMemcpyHostToDevice);
                        in_data[i].m = d_m;
                        break;
		case 'n':
			hipMalloc((void**)&d_c, dataCPU[i].n * sizeof(int));
                        hipMemcpy(d_c,dataCPU[i].c, dataCPU[i].n * sizeof(int) ,hipMemcpyHostToDevice);
                        hipMalloc((void**)&d_m, dataCPU[i].n * sizeof(int));
                        hipMemcpy(d_m,dataCPU[i].m, dataCPU[i].n * sizeof(int) ,hipMemcpyHostToDevice);
                       	hipMalloc((void**)&d_t, (dataCPU[i].n+1) *(dataCPU[i].k+1) * sizeof(int));
                        hipMemcpy(d_t,dataCPU[i].t, (dataCPU[i].n+1) *(dataCPU[i].k+1) *  sizeof(int),hipMemcpyHostToDevice);
			in_data[i].m = d_m;
			in_data[i].t = d_t;
                        break;
		case 's':
                        hipMalloc((void**)&d_c, dataCPU[i].n * sizeof(int));
                        hipMemcpy(d_c,dataCPU[i].c, dataCPU[i].n * sizeof(int) ,hipMemcpyHostToDevice);
                        hipMalloc((void**)&d_s, (dataCPU[i].n+1) *(dataCPU[i].k+1) * sizeof(bool));
                        hipMemcpy(d_s,dataCPU[i].s, (dataCPU[i].n+1) *(dataCPU[i].k+1) *  sizeof(bool),hipMemcpyHostToDevice);
                        in_data[i].s = d_s;
                        break;
		case 'l':
		case 'e':
			hipMalloc((void**)&d_x, dataCPU[i].n * sizeof(char));
                        hipMemcpy(d_x,dataCPU[i].x, dataCPU[i].n * sizeof(char) ,hipMemcpyHostToDevice);
                        hipMalloc((void**)&d_y, (dataCPU[i].k) * sizeof(char));
                        hipMemcpy(d_y,dataCPU[i].y, (dataCPU[i].k) *  sizeof(char),hipMemcpyHostToDevice);
			hipMalloc((void**)&d_m, (dataCPU[i].n+1) *(dataCPU[i].k+1) * sizeof(int));
                        hipMemcpy(d_m,dataCPU[i].m, (dataCPU[i].n+1) *(dataCPU[i].k+1) * sizeof(int) ,hipMemcpyHostToDevice);
                        in_data[i].x = d_x;
			in_data[i].y = d_y;
  			in_data[i].m = d_m;
			break;
                default:
                        break;
	}
	in_data[i].n = dataCPU[i].n;
        in_data[i].k = dataCPU[i].k;
        in_data[i].c = d_c;
        in_data[i].action = dataCPU[i].action;
	out_data[i].result = dataOut[i].result;
    }
    hipMalloc ( (void**) &dataGPUIn, input_size);
    hipMemcpy(dataGPUIn, in_data, input_size, hipMemcpyHostToDevice );
    hipMalloc ( (void**) &dataGPUOut, output_size);
    hipMemcpy(dataGPUOut, out_data, output_size, hipMemcpyHostToDevice );
    myKernel << <blocksPerGrid, threadsPerBlock >> >(dataGPUIn, dataGPUOut); // excute on kernel
    hipMemcpy(out_data, dataGPUOut, output_size, hipMemcpyDeviceToHost);
    i=0;
    for(; i<NUMBEROFTEST;i++){
	 switch (dataCPU[i].action) {
        	case 'c':
		case 'm':
		case 'f':
		case 'k':
		case 'n':
		case 's':
		case 'l':
		case 'e':
		case 'g':
		case 'o':
		case 'i':
			dataOut[i].result = out_data[i].result;
            		break;
   		default:
            		break;
    	}
    }
    hipFree(dataGPUIn);
    hipFree(dataGPUOut);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    time = time/1000;
    writeFile(time,fileName);
}
//************************************* Writing time in a FILE ******************************************

int main(int argc, char* argv[]) {
    inputData *dataCPU = (inputData*)malloc(sizeof(inputData)* NUMBEROFTEST);
    outputData *dataOut = (outputData*)malloc(sizeof(outputData)* NUMBEROFTEST);
    readingInput(dataCPU,dataOut);
    launch(dataCPU,dataOut,argv[1]);
    int index=0;
    for(;index< NUMBEROFTEST; index++) {
        printf("ID# %d ", index);
	switch (dataCPU[index].action) {
        	case 'c':
			printf("Value of C(%d, %d) is %d ", dataCPU[index].n, dataCPU[index].k, dataOut[index].result);
			break;
		case 'm':
			printf("Minimum number of multiplications is %d ",dataOut[index].result);
                        break;
		case 'f':
			printf("fibonacci %d",dataOut[index].result);
			break;
		case 'k':
			printf("Kane %d",dataOut[index].result);
			break;
		case 'n':
			printf("Max value %d",dataOut[index].result);
			break;
		case 's':
			if (dataOut[index].result == 1)
    				printf("Found a subset with sum");
 			 else
				 printf("No subset found with given sum");
			break;
		case 'l':
			printf("Length of LCS is %d",dataOut[index].result);
			break;
		case 'e':
			printf("Edit Distance is %d",dataOut[index].result);
                        break;
		case 'g':
			printf ("Minimum number of trials in worst case is %d",dataOut[index].result);
                        break;
		case 'o':
			printf("Optimal Game Stratgey %d",dataOut[index].result);
			break;
		case 'i':
			printf("Longest Increasing Subsequence Length is %d",dataOut[index].result);
			break;
		default:
			break;
		}
	printf("\n");
    }
    return 0;
}
